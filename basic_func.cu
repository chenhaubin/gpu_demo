#include "hip/hip_runtime.h"



__device__ bool f17(int i1)
{

		if (i1 % 4 == 0 && i1 % 100 != 0)
		{
			return true;
		}
		else
		{
			return false;
		}
}

__device__ int  f16(
	int i1,
	int i2,
	int i3
)
{
	if(i2 < 3)
	{
		--i1;
		i2 += 12;
	}
	return 365 * i1 + (i1 >> 2) - i1 / 100 + i1 / 400 + (153 * i2 - 457) / 5 + i3 -306;
}

__device__ int  f14(
	int* i1,
	int i2,
	int i3,
	int i4
)
{
	int t1 = i4 / 10000;
	int t2 = (i4  % 10000) / 100;
	int t3 = i4 % 100;
	int t4 = i1[0] / 10000;
	int t5 = (i1[0] % 10000) / 100;
	int t6 = i1[0] % 100;
	int t7 = f16(t1, t2, t3) - f16(t4, t5,t6);
	if(t7 > i3 || t7 < i2)
	{
		int t8 = t7 - (t7 /7 *2);
		if(t8 > i3)
		{
			t8 = i3;
		}
		while (i2 <= t8)
		{
			int t9 = i2 - (i2 - t8) / 2;
			if (i1[t9] > i4)
			{
				t8 = t9 - 1;
			}
			else
			{
				return t9;
			}
		}
	}
	else if(i1[t7] == i4)
	{
		return t7;
	}
	
	else
	{
		if(i1[t7] < i4)
		{
			i2 = t7;
		}
		else if(i1[t7] > i4)
		{
			i3 = t7;
		}
		while (i2 <= i3)
		{
			int t9 = i2 - (i2 - i3) / 2; 
			if (i1[t9] > i4)
			{
				i3 = t9 - 1;
			}
			else
			{
				return t9;
			}
		}
	
	}
	return -1;
}

__device__ int f15(
	int i1,
	int* i2,
	bool* b1,
	int i3,
	bool* b2
)
{
	if (0 == i3) return -1;
	int t1 = f14(
		i2,
		0,
		i3 - 1,
		i1
	);
	
	if (-1 == t1)
	{
		return -1;
	}
	*b2 = b1[t1];
	return 0;
}

__device__ int f13(
	int i1,
	int* i2
)
{

	*i2 = -1;
	int t1 = i1 % 100;
	int t2 = i1 % 10000 / 100;
	int t3 = i1 / 10000;

		if (1 == t2 || 3 == t2 || 5 == t2 || 7 == t2 ||
			8 == t2 || 10 == t2 || 12 == t2)
		{
			if (t3 > 31)
			{
				*i2 = (i1 / 100) * 100 + 31;
			}
			else
			{
				*i2 = i1;
			}

		}
		else if (2 != t2)
		{
			if (t3 > 30)
			{
				*i2 = (i1 / 100) * 100 + 30;
			}
			else
			{
				*i2 = i1;
			}

		}
		else {
				*i2 = (i1 / 100) * 100 + 28;
		}
	
	return 0;
}

__device__ int f12(
	int i1,
	int i2,
	int* i3,
	int i4,
	int* i5
)
{

	if (0 == i4) return -1;
	int t1 = f14(i3, 0, i4 - 1, i1);
	int t2 = f14(i3, 0, i4 - 1, i2);
	if (-1 == t1 || -1 == t2)
	{
		return -1;
	}
	*i5 = t2 - t1;

	return 0;
}

__device__ int f9(
	int i1,
	int i2,
	int* i3,
	bool* b1,
	int i4,
	double* d1
)
{
	int t1 = 0;
	*d1 = 0;
    f12(
        i1,
        i2,
        i3,
        i4,
        &t1
		);
    *d1 = (double)(t1) / 365;
	return 0;
}
__device__ int  f10(
	int i1,
	int i2,
	int* i3
)
{
	int t1 = i1 / 10000;
	int t2 = (i1 / 100) % 100;
	int t3 = i1 % 100;
	*i3 = (t1 + (t2 + i2 - 1) / 12) * 10000 + (((t2 + i2 - 1) % 12) + 1) * 100 + t3;
	f13(*i3, i3);

	return 0;
}

__device__ int f11(
	int i1,
	int* i2
)
{

	*i2 = -1;
	int t1 = i1 % 10000 / 100;
	int t2 = i1 / 10000;
	if (1 == t1 || 3 == t1 || 5 == t1 || 7 == t1 ||
		8 == t1 || 10 == t1 || 12 == t1)
	{
		*i2 = (i1 / 100) * 100 + 31;
	}
	else if (2 != t1)
	{
		*i2 = (i1 / 100) * 100 + 30;
	}
	else {
			*i2 = (i1 / 100) * 100 + 28;
	}
	return 0;
}

__device__ int f7(
	int i1,
	int i2,
	int i3,
	int* i4,
	int i5,
	int* i6
)
{
	int t1 = 0;
	int t2 = 0;
	int t3 = 0;

    f10(
        i1,
        i2,
        &t3
    );
    
    f11(i1, &t2);
    if (i1 == t2) {
        f11(t3, i6);
    }
    else {
        *i6 = t3;
    }

	return 0;
}

__device__ int f8(
	int i1,
	int* i2,
	bool* b1,
	int i3,
	int* i4
)
{
	bool t1 = false;
	f15(
		i1,
		i2,
		b1,
		i3,
		&t1);
	if (true == t1)
	{
		*i4 = i1;
		return 0;
	}
}

__device__ int f6(
	int i1,
	int i2,
	int i3,
	int* i4,
	bool* b1,
	int i5,
	int* i6
)
{
	f7(
		i1,
		i2,
		i3,
		i4,
		i5,
		i6
	);
	f8(
		*i6,
		i4,
		b1,
		i5,
		i6
	);
	return 0;
}

__device__  int f2(
	int i1,
	int i2,
	int i3,
	int i4,
	int i5,
	int i6,
	int i7,
	int* i8,
	bool* b1,
	int i9,
	int i10,
	int* i11,
	int* i12
)
{
    int t1 = 0;
    int t2 = 0;
    int t3 = i4;
    int t4 = t3;

    
    t2 = 0;
    t1 = i1;
    i11[0] = i2;
    t2 = 1;
    t1 += 1;
        

    while (t4 < i5)
    {
        f6(
            i4,
            i6 * t1,
            i7,
            i8,
            b1,
            i9,
            &t4
        );
        
        if (t4 >= i5)
        {
            i11[t2] = i5;
            ++t2;
            
            break;
        }
        else
        {
            i11[t2] = t4;
            ++t2;
            ++t1;
        }
    }
    if (i10 < t2)
    {
        return -1;
    }

    *i12 = t2;
	return 0;
}


__device__ double f3(
    double  d1,
    double  d2,
    int*    i1,
    int     i2,
    int*    i3,
    bool*   b1,
    int     i4
){
	double t1 = 0.0;
    double t2 =0.0;
	for (int i = 0; i < i2; ++i)
	{
		f9(
			i1[i],
			i1[i + 1],
			i3,
			b1,
			i4,
			&t1
		);
        t2 += (d2 / 100.0 * t1 * d1);
	}

	return t2;
}

__device__ int  f4(
	int i1,
	int i2,
	int* i3,
	int i4,
	int* i5
)
{
	if (0 == i4) return -1;
	if (i1 > i2) return -1;
	int t1 = i1 / 10000;
	int t2 = i1 % 10000 / 100;
	int t3 = i2 / 10000;
	int t4 = i2 % 10000 / 100;
	*i5 = (t3 - t1) * 12 + t4 - t2;
	return 0;
}

__device__  int f5(
	int i1,
	int i2,
	int i3,
	int i4,
	int i5,
	int* i6,
	bool* b1,
	int i7
)
{
	int t1 = 0;
	int t2 = 0;
	if (i1 > i3)
	{
		return -1;
	}
	if (i2 <= i3)
	{
		return -1;
	}
	int t3 = 0;
	int t4 = 0;
    f4(
        i1,
        i3,
        i6,
        i7,
        &t3
    );
    t1 = (int)floor((double)t3 / (double)i4);

    f7(
        i1,
        t1 * i4,
        i5,
        i6,
        i7,
        &t2
    );
    
    f8(
         t2,
         i6,
         b1,
         i7,
         &t2
    );
    
    if (t2 > i3)
    {
        t1 -= 1;
    }

	return t1;
}

__device__  int f1(
	int i1,
	int i2,
	int i3,
	int i4,
	int i5,
	int* i7,
	bool* b1,
	int i8,
	int* i9,
	bool* b2,
	bool* b3,
	int* i11,
	int* i12
)
{
	*b3 = false;
	*b2 = false;
	int t1 = 0;
	bool t2 = true;

	int t3 = 0;

	int t4 = i2;
		t1 = f5(
			i2,
			i3,
			i1,
			i4,
			i5,
			i7,
			b1,
			i8
		);

			*b2 = false;
			while (t3 < i3)
			{
				
                f6(
                    i2,
                    i4 * t1,
                    i5,
                    i7,
                    b1,
                    i8,
                    &t3
                );
				
				if (t3 > i1 && t2 == true)
				{
					t2 = false;
					*i9 = t1 - 1;
					
						f6(
							i2,
							i4 * (*i9),
							i5,
							i7,
							b1,
							i8,
							i11
						);
					
					if (t3 >= i3)
					{
						*b3 = true;
						*i12 = i3;
					}
					else
					{
						*i12 = t3;
					}
					break;
				}
				t4 = t3;
				++t1;
			}
		

	
	return 0;
}
