#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "basic_func.h"
#include <stdlib.h>
#include <stdio.h>
__device__ double single_calc(
    int i1, 
    int i2, 
    int i3,
    int i4, 
    int i5, 
    int i6, 
    double d1, 
    double d2,
    int * i7, 
    bool *b1, 
    int i8){

    int t1 = 0;
    bool t2 = false;
    bool t3 = false;
    int t4 = i3;
    int t5 = i4;


    f1(
            i1,
            i3,
            i4,
            i5,
            i6,
            i7,
            b1,
            i8,
            &t1,
            &t2,
            &t3,
            &t4,
            &t5
        );
    
    int t6 = 0;
    int * t7 = (int*)malloc(sizeof(int) * (i2 + 1));
    f2(
        t1,
        t4,
        t5,
        i3,
        i4,
        i5,
        i6,
        i7,
        b1,
        i8,
        i2 + 1,
        t7,
        &t6
    );
    double t8 = f3(
        d2,
        d1,
        t7,
        t6,
        i7,
        b1,
        i8
        );
    
    free(t7);
    return t8;
}

__global__  void   calc_ken(
    int i1, 
    int * i2, 
    int * i3,
    int * i4, 
    int * i5, 
    int * i6,
    double * d1, 
    double * d2, 
    int * i7, 
    bool *b1, 
    int i8,
    double *d4
){
    int c = blockIdx.x;
    d4[c]= single_calc(
        i1, 
        i2[c], 
        i3[c],
        i4[c], 
        i5[c], 
        i6[c], 
        d1[c], 
        d2[c], 
        i7, 
        b1, 
        i8);
}

extern "C" int  batch_calc(
    int i1, 
    int i2,
    int * i3, 
    int * i4,
    int * i5, 
    int * i6, 
    int * i7,
    double * d1, 
    double * d2, 
    int * i8, 
    bool *b1, 
    int i9,
    double *d3
    ){
    
    int * dev_i3=NULL;
	int * dev_i4=NULL;
	int * dev_i5=NULL;
	int * dev_i6=NULL;
	int * dev_i7=NULL; 
	double * dev_d1    =NULL;
	double * dev_d2    =NULL;
    int *dev_i8    =NULL;
    bool* dev_b1 =NULL;
    double * dev_d3 = NULL;
    hipError_t cudaStatus;
    
    cudaStatus  = hipMalloc((void**)&dev_i3, i2 * sizeof(int));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_i3\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_i3, i3, i2  * sizeof(int), hipMemcpyHostToDevice);
    
    
    cudaStatus  = hipMalloc((void**)&dev_i4, i2 * sizeof(int));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_i4\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_i4, i4, i2  * sizeof(int), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_i5, i2 * sizeof(int));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_i5\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_i5, i5, i2  * sizeof(int), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_i6, i2 * sizeof(int));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_i6\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_i6, i6, i2  * sizeof(int), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_i7, i2 * sizeof(int));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_i7\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_i7, i7, i2  * sizeof(int), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_d1, i2 * sizeof(double));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_d1\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_d1, d1, i2  * sizeof(double), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_d2, i2 * sizeof(double));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_d2\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_d2, d2, i2  * sizeof(double), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_i8, i9 * sizeof(int));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_i8\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_i8, i8, i9  * sizeof(int), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_b1, i9 * sizeof(bool));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_b1\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemcpy(dev_b1, b1, i9  * sizeof(bool), hipMemcpyHostToDevice);
    
    cudaStatus  = hipMalloc((void**)&dev_d3, i2 * sizeof(double));                   
    if(cudaStatus != hipSuccess )                                                                         
    {                                                                                                      
        printf("\ncudaMalloc CU_FAILed!<<<%s>>>\n",hipGetErrorString(cudaStatus));                        
        printf("Error:<%s,%s,%d>  dev_d4\n",__FILE__,__FUNCTION__,__LINE__ );               
        goto Error;                                                                                        
    }                                                                                                      
    hipMemset(dev_d3, 0x00, i2  * sizeof(double));
    
    calc_ken<<< i2,1>>>(
        i1, 
        dev_i3, 
        dev_i4,
        dev_i5, 
        dev_i6, 
        dev_i7, 
        dev_d1, 
        dev_d2, 
        dev_i8, 
        dev_b1, 
        i9,
        dev_d3);
    
    hipMemcpy(d3, dev_d3, i2 * sizeof(double), hipMemcpyDeviceToHost);
Error:
    hipFree(dev_i3);
    hipFree(dev_i4);
    hipFree(dev_i5);
    hipFree(dev_i6);
    hipFree(dev_i7);
    hipFree(dev_d1);
    
    hipFree(dev_d2);
    hipFree(dev_d3);
    hipFree(dev_i8);
    
    hipFree(dev_b1);
    return 0;
}







